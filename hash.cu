#include <stdlib.h>
#include <stdio.h>
#include <string>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void HashingKernel(int *cuda_hash_table,int *cuda_a_list, int *cuda_b_list, int *cuda_random_value,int* cuda_func_index, int n, int p,int *cuda_kicked_list,int t){
	int key = threadIdx.x + blockDim.x*blockIdx.x;
	int index = cuda_func_index[key];
	int hash_value = ((cuda_a_list[index] * cuda_random_value[key] + cuda_b_list[index]) % p) % n;
	// if (key==16777217)
	// 	printf("%d,%d\n",cuda_a_list[index] * cuda_random_value[key] + cuda_b_list[index],hash_value);
	if (cuda_kicked_list[key]==1 || cuda_hash_table[hash_value]==0){
		
		cuda_hash_table[hash_value] = cuda_random_value[key];
		cuda_func_index[key] = (cuda_func_index[key] + 1) % t;
	}
}
__global__ void CheckKickKernel(int *cuda_hash_table,int *cuda_a_list, int *cuda_b_list, int *cuda_random_value,int* cuda_func_index, int n, int p,int *cuda_kicked_list,int t){
	int key = threadIdx.x + blockDim.x*blockIdx.x;
	int index = (cuda_func_index[key]+t-1)%t;
	int hash_value = ((cuda_a_list[index] * cuda_random_value[key] + cuda_b_list[index]) % p) % n;

	if (cuda_hash_table[hash_value]==cuda_random_value[key]){
		cuda_kicked_list[key] = 0;
	}else{
		cuda_kicked_list[key] = 1;
	}
}

int main(){
	int i;
	int input_size;
	int s;
	int t;
	int n;
	int p;
	int bound_length;
	int *a_list;
	int *b_list;
	int *random_value;
	int *kicked_list;
	int *hash_table;
	int *func_index;	
	int *cuda_a_list;
	int *cuda_b_list;
	int *cuda_random_value;
	int *cuda_hash_table;
	int *cuda_kicked_list;
	int *cuda_func_index;
	int block_num;
	int block_size;
	int sum;


	s = 20;
	input_size = pow(2,s);
	t = 2;
	n = pow(2,25);
	p = 75000103;
	bound_length = (int)4*log(n);
	block_num = input_size/256;
	// printf("block_num=%d\n", block_num);
	block_size = 256;

	// printf("input_size=%d\n", input_size);
	// printf("n=%d\n",n);

	hipMalloc((void **) &cuda_a_list,sizeof(int)*t);
	hipMalloc((void **) &cuda_b_list,sizeof(int)*t);
	hipMalloc((void **) &cuda_hash_table,sizeof(int)*n);
	hipMalloc((void **) &cuda_random_value,sizeof(int)*input_size);
	hipMalloc((void **) &cuda_kicked_list,sizeof(int)*input_size);
	hipMalloc((void **) &cuda_func_index,sizeof(int)*input_size);

	a_list = (int*)malloc(sizeof(int)*t);
	b_list = (int*)malloc(sizeof(int)*t);
	hash_table = (int*)malloc(sizeof(int)*n);
	random_value = (int*)malloc(sizeof(int)*input_size);
	kicked_list = (int*)malloc(sizeof(int)*input_size);
	func_index = (int*)malloc(sizeof(int)*input_size);

	srand(time(0));
	for (i = 0;i < t;i++){
		a_list[i] = rand()%100;
		b_list[i] = rand()%100;
		while (a_list[i]==0){
			a_list[i]=rand()%100;
		}
		while (b_list[i]==0){
			b_list[i]=rand()%100;
		}
	}
	for (i = 0;i < input_size;i++){
		random_value[i] = rand()%1000000;
		while (random_value[i]==0) {
			random_value[i] = rand()%1000000;
		}
		// printf("random_value[%d]=%d\n",i,random_value[i] );
	}
	memset(hash_table,0,sizeof(int)*n);
	memset(kicked_list,0,sizeof(int)*input_size);
	memset(func_index,0,sizeof(int)*input_size);

	hipMemcpy(cuda_a_list,a_list,sizeof(int)*t,hipMemcpyHostToDevice);
	hipMemcpy(cuda_b_list,b_list,sizeof(int)*t,hipMemcpyHostToDevice);
	hipMemcpy(cuda_hash_table,hash_table,sizeof(int)*n,hipMemcpyHostToDevice);
	hipMemcpy(cuda_random_value,random_value,sizeof(int)*input_size,hipMemcpyHostToDevice);
	hipMemcpy(cuda_kicked_list,kicked_list,sizeof(int)*input_size,hipMemcpyHostToDevice);
	hipMemcpy(cuda_func_index,func_index,sizeof(int)*input_size,hipMemcpyHostToDevice);

	int count = 0;
	while(1){
		sum = 0;
		HashingKernel<<<block_num,block_size>>>(cuda_hash_table,cuda_a_list,cuda_b_list,cuda_random_value,cuda_func_index,n,p,cuda_kicked_list,t);
		// cudaMemcpy(hash_table,cuda_hash_table,sizeof(int)*n,cudaMemcpyDeviceToHost);
		// for (i=0;i<n;i++){
		// 	// printf("hash_table[%d]=%d\n",i,hash_table[i] );
		// 	if (hash_table[i]!=0){
		// 		printf("there is conflict\n");
		// 	}
		// }
		// break;
		CheckKickKernel<<<block_num,block_size>>>(cuda_hash_table,cuda_a_list,cuda_b_list,cuda_random_value,cuda_func_index,n,p,cuda_kicked_list,t);
		hipMemcpy(kicked_list,cuda_kicked_list,sizeof(int)*input_size,hipMemcpyDeviceToHost);
		for (i = 0;i<input_size;i++){
			sum+=kicked_list[i];
		}
		printf("sum = %d\n",sum );
		if (sum == 0){
			break;
		}
		count += 1;
		if(count == bound_length){
			count = 0;
			printf("------------------------Restart!------------------------\n");
			for (i = 0;i < t;i++){
				a_list[i] = rand()%1000;
				b_list[i] = rand()%1000;
				while (a_list[i]==0){
					a_list[i]=rand()%1000;
				}
				while (b_list[i]==0){
					b_list[i]=rand()%1000;
				}
			}
			memset(hash_table,0,sizeof(int)*n);
			memset(kicked_list,0,sizeof(int)*input_size);
			memset(func_index,0,sizeof(int)*input_size);

			hipMemcpy(cuda_a_list,a_list,sizeof(int)*t,hipMemcpyHostToDevice);
			hipMemcpy(cuda_b_list,b_list,sizeof(int)*t,hipMemcpyHostToDevice);
			hipMemcpy(cuda_hash_table,hash_table,sizeof(int)*n,hipMemcpyHostToDevice);
			hipMemcpy(cuda_kicked_list,kicked_list,sizeof(int)*input_size,hipMemcpyHostToDevice);
			hipMemcpy(cuda_func_index,func_index,sizeof(int)*input_size,hipMemcpyHostToDevice);
		}
	}

	hipMemcpy(hash_table,cuda_hash_table,sizeof(int)*n,hipMemcpyDeviceToHost);
	hipMemcpy(func_index,cuda_func_index,sizeof(int)*input_size,hipMemcpyDeviceToHost);
	int index = (func_index[0]+t-1)%t;
	printf("index=%d\n", index);
	int hash_value=((a_list[index] * random_value[0] + b_list[index]) % p) % n;
	printf("hash_value=%d\n",hash_value );
	if(hash_table[hash_value]==random_value[0])
		printf("Rigth anwser!\n");

	free(a_list);
	free(b_list);
	free(hash_table);
	free(random_value);
	free(func_index);
	free(kicked_list);
	hipFree(cuda_a_list);
	hipFree(cuda_b_list);
	hipFree(cuda_hash_table);
	hipFree(cuda_random_value);
	hipFree(cuda_func_index);
	hipFree(cuda_kicked_list);
	return 0;
}
