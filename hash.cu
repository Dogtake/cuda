#include <stdlib.h>
#include <stdio.h>
#include <string>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void HashingKernel(int *cuda_hash_table,int *cuda_a_list, int *cuda_b_list, int cuda_p, int *random_value){
	int key = threadIdx.x + blockDim.x*blockIdx.x;
	int value = random_value[key];
	//printf("key = %d, value = %d\n",key,value);

}

int main(){
	int i;
	int input_size;
	int s;
	int t;
	int n;
	int p;
	int bound_length;
	int *a_list;
	int *b_list;
	int *random_value;
	int *hash_table;	
	int *cuda_a_list;
	int *cuda_b_list;
	int *cuda_random_value;
	int *cuda_hash_table;
	int block_num;
	int block_size;

	s = 10;
	input_size = pow(2,s);
	t = 2;
	n = pow(2,25);
	p = 75000103;
	bound_length = (int)4*log(n);
	block_num = input_size/256;
	block_size = 256;
	printf("input_size = %d\n",input_size);

	hipMalloc((void **) &cuda_a_list,sizeof(int)*t);
	hipMalloc((void **) &cuda_b_list,sizeof(int)*t);
	hipMalloc((void **) &cuda_hash_table,sizeof(int)*n);
	hipMalloc((void **) &cuda_random_value,sizeof(int)*input_size);

	a_list = (int*)malloc(sizeof(int)*t);
	b_list = (int*)malloc(sizeof(int)*t);
	hash_table = (int*)malloc(sizeof(int)*n);
	random_value = (int*)malloc(sizeof(int)*input_size);

	srand(0);
	for (i = 0;i < t;i++){
		a_list[i] = rand()%1000;
		b_list[i] = rand()%1000;
		//printf("%d\n",a_list[i]);
	}
	for (i = 0;i < input_size;i++){
		random_value[i] = rand();
		//printf("%d\n",random_value[i]);
	}

	hipMemcpy(cuda_a_list,a_list,sizeof(int)*t,hipMemcpyHostToDevice);
	hipMemcpy(cuda_b_list,b_list,sizeof(int)*t,hipMemcpyHostToDevice);
	hipMemcpy(cuda_random_value,random_value,sizeof(int)*input_size,hipMemcpyHostToDevice);


	HashingKernel<<<block_num,block_size>>>(cuda_hash_table,cuda_a_list,cuda_b_list,p,cuda_random_value);

	hipMemcpy(hash_table,cuda_hash_table,sizeof(int)*n,hipMemcpyDeviceToHost);

	free(a_list);
	free(b_list);
	free(hash_table);
	free(random_value);
	hipFree(cuda_a_list);
	hipFree(cuda_b_list);
	hipFree(cuda_hash_table);
	hipFree(cuda_random_value);
	return 0;
}
