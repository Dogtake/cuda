#include <stdlib.h>
#include <stdio.h>
#include <string>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void HashingKernel(int key,int value){
}

int main(){
	int i;
	int input_size;
	int s;
	int t;
	int n;
	int p;
	int bound_length;
	int *a_list;
	int *b_list;
	int *hash_table;	
	int *cuda_a_list;
	int *cuda_b_list;
	int *cuda_hash_table;

	s = 10;
	input_size = 2^s;
	t = 2;
	n = 2^25;
	p = 218641;
	bound_length = 4*log(n);

	hipMalloc((void **) &cuda_a_list,sizeof(int)*t);
	hipMalloc((void **) &cuda_b_list,sizeof(int)*t);
	hipMalloc((void **) &cuda_hash_table,sizeof(int)*n);

	a_list = (int*)malloc(sizeof(int)*t);
	b_list = (int*)malloc(sizeof(int)*t);
	hash_table = (int*)malloc(sizeof(int)*n);

	for (i = 0;i < t;i++){
		a_list[i] = rand();
		b_list[i] = rand();
	}

	hipMemcpy(cuda_a_list,a_list,sizeof(int)*t,hipMemcpyHostToDevice);
	hipMemcpy(cuda_b_list,b_list,sizeof(int)*t,hipMemcpyHostToDevice);


	free(a_list);
	free(b_list);
	free(hash_table);
	hipFree(cuda_a_list);
	hipFree(cuda_b_list);
	hipFree(cuda_hash_table);
	return 0;
}
