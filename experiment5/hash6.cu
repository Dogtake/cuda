#include <stdlib.h>
#include <stdio.h>
#include <string>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void HashingKernel(int *cuda_hash_table,int *cuda_a_list, int *cuda_b_list, int *cuda_random_value,int* cuda_func_index, int n, int p,int *cuda_kicked_list,int t,int flag){
	int key = threadIdx.x + blockDim.x*blockIdx.x;
	int index = cuda_func_index[key];
	unsigned int hash_value = (unsigned(cuda_a_list[index] * cuda_random_value[key] * cuda_random_value[key] + cuda_b_list[index]* cuda_random_value[key]* +cuda_b_list[index]) % p) % n;
	// unsigned int hash_value = (unsigned(cuda_a_list[index] * cuda_random_value[key] +cuda_b_list[index]) % p) % n;
	// printf("hash_value=%d\n",hash_value );
	if (cuda_kicked_list[key]==1 || flag==0){
		
		cuda_hash_table[hash_value] = cuda_random_value[key];
		cuda_func_index[key] = (cuda_func_index[key] + 1) % t;
		
	}
}
__global__ void CheckKickKernel(int *cuda_hash_table,int *cuda_a_list, int *cuda_b_list, int *cuda_random_value,int* cuda_func_index, int n, int p,int *cuda_kicked_list,int t){
	int key = threadIdx.x + blockDim.x*blockIdx.x;
	int index = (cuda_func_index[key]+t-1)%t;
	unsigned int hash_value = (unsigned(cuda_a_list[index] * cuda_random_value[key] * cuda_random_value[key] + cuda_b_list[index]* cuda_random_value[key]* +cuda_b_list[index]) % p) % n;
	// unsigned int hash_value = (unsigned(cuda_a_list[index] * cuda_random_value[key] +cuda_b_list[index]) % p) % n;
	if (cuda_hash_table[hash_value]==cuda_random_value[key]){
		cuda_kicked_list[key] = 0;
	}else{
		cuda_kicked_list[key] = 1;
	}
}

__global__ void LookUpKernel(int *cuda_hash_table,int *cuda_a_list,int *cuda_b_list, int t,int n,int p, int *cuda_lookup_table,int *cuda_results){
	int key = threadIdx.x + blockDim.x*blockIdx.x;
	int i,hash_value;

	for(i = 0;i < t;i++){
		hash_value = (unsigned(cuda_a_list[i] * unsigned(cuda_lookup_table[key]) + cuda_b_list[i]) % p) % n;
		if (cuda_hash_table[hash_value] == cuda_lookup_table[key]){
			cuda_results[key] = 1;
			break;
		}
	}
}

int main(int argc,char const *argv[]){
	int i;
	unsigned int input_size;
	unsigned int s;
	int t;
	unsigned int n;
	unsigned int p;
	unsigned int bound_length;
	int *a_list;
	int *b_list;
	int *random_value;
	int *kicked_list;
	int *hash_table;
	int *func_index;	
	int *cuda_a_list;
	int *cuda_b_list;
	int *cuda_random_value;
	int *cuda_hash_table;
	int *cuda_kicked_list;
	int *cuda_func_index;
	int block_num;
	int block_size;
	int sum;
	int flag;
	int len;
	float ts;
	clock_t start,end;
	

	s = atoi(argv[1]);
	t = atoi(argv[2]);
	ts = (float)strtod(argv[4],NULL);
	len = atoi(argv[5]);

	input_size = pow(2,s);
	if (s == 24 ){
		if (t==2){
			input_size-=pow(2,22);
		}else if (t == 3){
			input_size-=pow(2,15);
		}
	}
	n = (int)(input_size*ts);
	p = 85000173;
	bound_length = len*(int)log(n);
	block_num = input_size/256;
	block_size = 256;


	hipMalloc((void **) &cuda_a_list,sizeof(int)*t);
	hipMalloc((void **) &cuda_b_list,sizeof(int)*t);
	hipMalloc((void **) &cuda_hash_table,sizeof(int)*n);
	hipMalloc((void **) &cuda_random_value,sizeof(int)*input_size);
	hipMalloc((void **) &cuda_kicked_list,sizeof(int)*input_size);
	hipMalloc((void **) &cuda_func_index,sizeof(int)*input_size);

	a_list = (int*)malloc(sizeof(int)*t);
	b_list = (int*)malloc(sizeof(int)*t);
	hash_table = (int*)malloc(sizeof(int)*n);
	random_value = (int*)malloc(sizeof(int)*input_size);
	kicked_list = (int*)malloc(sizeof(int)*input_size);
	func_index = (int*)malloc(sizeof(int)*input_size);

	srand(time(0));
	for (i = 0;i < t;i++){
		a_list[i] = (unsigned)rand()%10000;
		b_list[i] = (unsigned)rand()%10000;
		while (a_list[i]==0){
			a_list[i]=(unsigned)rand();
		}
		while (b_list[i]==0){
			b_list[i]=(unsigned)rand();
		}
	}
	for (i = 0;i < input_size;i++){
		random_value[i] =unsigned( rand());
		while (random_value[i]==0) {
			random_value[i] = unsigned(rand());
		}
	}
	memset(hash_table,0,sizeof(int)*n);
	memset(kicked_list,0,sizeof(int)*input_size);
	memset(func_index,0,sizeof(int)*input_size);

	hipMemcpy(cuda_a_list,a_list,sizeof(int)*t,hipMemcpyHostToDevice);
	hipMemcpy(cuda_b_list,b_list,sizeof(int)*t,hipMemcpyHostToDevice);
	hipMemcpy(cuda_hash_table,hash_table,sizeof(int)*n,hipMemcpyHostToDevice);
	hipMemcpy(cuda_random_value,random_value,sizeof(int)*input_size,hipMemcpyHostToDevice);
	hipMemcpy(cuda_kicked_list,kicked_list,sizeof(int)*input_size,hipMemcpyHostToDevice);
	hipMemcpy(cuda_func_index,func_index,sizeof(int)*input_size,hipMemcpyHostToDevice);

	int count = 0;
	int base = pow(2,24);
	start=clock();
	int first = 0;
	while(1){
		if (first == 0){
			flag = 0;
		}else{
			flag =1;
		}
		sum = 0;
		first = 1;
		HashingKernel<<<block_num,block_size>>>(cuda_hash_table,cuda_a_list,cuda_b_list,cuda_random_value,cuda_func_index,n,p,cuda_kicked_list,t,flag);
		CheckKickKernel<<<block_num,block_size>>>(cuda_hash_table,cuda_a_list,cuda_b_list,cuda_random_value,cuda_func_index,n,p,cuda_kicked_list,t);
		hipMemcpy(kicked_list,cuda_kicked_list,sizeof(int)*input_size,hipMemcpyDeviceToHost);
		for (i = 0;i<input_size;i++){
			sum+=kicked_list[i];
		}
		// printf("sum=%d,base=%d\n",sum,base);
		if(sum < base){
			count = 0;
			base = sum;
		}else{
			count += 1;
		}
		// printf("base = %d\n",base );
		if (sum == 0){
			break;
		}
		if(count > bound_length){
			count = 0;
			first = 0;
			// printf("------------------------Restart!------------------------\n");
			base = pow(2,24);
			for (i = 0;i < t;i++){
				a_list[i] = rand();
				b_list[i] = rand();
				while (a_list[i]==0){
					a_list[i]=rand();
				}
				while (b_list[i]==0){
					b_list[i]=rand();
				}
			}
			memset(hash_table,0,sizeof(int)*n);
			memset(kicked_list,0,sizeof(int)*input_size);
			memset(func_index,0,sizeof(int)*input_size);

			hipMemcpy(cuda_a_list,a_list,sizeof(int)*t,hipMemcpyHostToDevice);
			hipMemcpy(cuda_b_list,b_list,sizeof(int)*t,hipMemcpyHostToDevice);
			hipMemcpy(cuda_hash_table,hash_table,sizeof(int)*n,hipMemcpyHostToDevice);
			hipMemcpy(cuda_kicked_list,kicked_list,sizeof(int)*input_size,hipMemcpyHostToDevice);
			hipMemcpy(cuda_func_index,func_index,sizeof(int)*input_size,hipMemcpyHostToDevice);
		}
	}
	end=clock();	
	hipMemcpy(hash_table,cuda_hash_table,sizeof(int)*n,hipMemcpyDeviceToHost);
	hipMemcpy(func_index,cuda_func_index,sizeof(int)*input_size,hipMemcpyDeviceToHost);

	
	printf("%f\n",(double)(end-start)/CLOCKS_PER_SEC );
	//##########################################################################################
	// Experiment 2
	// printf("%d\n", input_size);
	if (argc == 4 && input_size>pow(2,23)){
		int counter;
		float percent = float(100-10*atoi(argv[3]))/100.0;
		int *results;
		int *lookup_table;
		int *cuda_results;
		int *cuda_lookup_table;

		// printf("Insertion Finished. Start Exp2:\n");
		// printf("percent=%f\n",percent );

		lookup_table  = (int*)malloc(sizeof(int)*input_size);
		results = (int *)malloc(sizeof(int)*input_size);
		memset(lookup_table,0,sizeof(int)*input_size);
		memset(results,0,sizeof(int)*input_size);
		
		hipMalloc((void **) &cuda_results,sizeof(int)*input_size);
		hipMalloc((void **) &cuda_lookup_table,sizeof(int)*input_size);

		for (i=0;i<input_size;i++){
			if (i<(int)(input_size*percent)){
				lookup_table[i] = random_value[rand()%input_size];
			}else{
				lookup_table[i] = rand();
			}
		}	
		hipMemcpy(cuda_results,results,sizeof(int)*input_size,hipMemcpyHostToDevice);
		hipMemcpy(cuda_lookup_table,lookup_table,sizeof(int)*input_size,hipMemcpyHostToDevice);
		
		counter = 0;

		start = clock();
		LookUpKernel<<<block_num,block_size>>>(cuda_hash_table,cuda_a_list,cuda_b_list,t,n,p,cuda_lookup_table,cuda_results);
		end = clock();

		hipMemcpy(results,cuda_results,sizeof(int)*input_size,hipMemcpyDeviceToHost);
		for(i =0;i<input_size;i++){
			counter +=  results[i];
		}
		if (counter>=(int)(input_size*percent)){
			// printf("counter = %d,percent = %d\n", counter,(int)(input_size*percent));
			printf("%f\n",(double)(end-start)/CLOCKS_PER_SEC);
		}
		
		free(lookup_table);
		free(results);
		hipFree(cuda_results);
		hipFree(cuda_lookup_table);
	}



	free(a_list);
	free(b_list);
	free(hash_table);
	free(random_value);
	free(func_index);
	free(kicked_list);
	hipFree(cuda_a_list);
	hipFree(cuda_b_list);
	hipFree(cuda_hash_table);
	hipFree(cuda_random_value);
	hipFree(cuda_func_index);
	hipFree(cuda_kicked_list);
	

	return 0;
}
